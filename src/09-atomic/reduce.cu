#include "hip/hip_runtime.h"
#include "error.cuh"
#include <stdio.h>

#ifdef USE_DP
    typedef double real;
#else
    typedef float real;
#endif

const int NUM_REPEATS = 100;

void timing(real *h_x, real *d_x, const int N, const bool atomic);
real reduce(real *d_x, const int N, const bool atomic);
template<bool using_atomic>
void __global__ reduce(real *d_x, real *d_y, const int N);

int main(void)
{
    const int N = 100000000;
    const int M = sizeof(real) * N;
    real *h_x = (real *) malloc(M);
    for (int n = 0; n < N; ++n)
    {
        h_x[n] = 1.23;
    }
    real *d_x;
    CHECK(hipMalloc(&d_x, M));

    printf("\nusing two kernels:\n");
    timing(h_x, d_x, N, false);
    printf("\nusing atomicAdd:\n");
    timing(h_x, d_x, N, true);

    free(h_x);
    CHECK(hipFree(d_x));
    return 0;
}

void timing(real *h_x, real *d_x, const int N, const bool atomic)
{
    real sum = 0;
    float t_sum = 0;
    float t2_sum = 0;

    for (int repeat = 0; repeat <= NUM_REPEATS; ++repeat)
    {
        const int M = sizeof(real) * N;
        CHECK(hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice));

        hipEvent_t start, stop;
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        CHECK(hipEventRecord(start));

        sum = reduce(d_x, N, atomic); 

        CHECK(hipEventRecord(stop));
        CHECK(hipEventSynchronize(stop));
        float elapsed_time;
        CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
        printf("Time = %g ms.\n", elapsed_time);

        if (repeat > 0)
        {
            t_sum += elapsed_time;
            t2_sum += elapsed_time * elapsed_time;
        }

        CHECK(hipEventDestroy(start));
        CHECK(hipEventDestroy(stop));
    }

    const float t_ave = t_sum / NUM_REPEATS;
    const float t_err = sqrt(t2_sum / NUM_REPEATS - t_ave * t_ave);
    printf("Time = %g +- %g ms.\n", t_ave, t_err);

    printf("sum = %f.\n", sum);
}

real reduce(real *d_x, const int N, const bool atomic)
{
    const int block_size = 1024;
    const int repeat_size = 10;
    int grid_size = (N + block_size - 1) / block_size;
    grid_size = (grid_size + repeat_size - 1) / repeat_size;
    const int ymem = atomic ? sizeof(real) : sizeof(real) * grid_size;
    const int smem = sizeof(real) * block_size;

    real h_y[1] = {0};
    real *d_y;
    CHECK(hipMalloc(&d_y, ymem));

    if (atomic)
    {
        CHECK(hipMemcpy(d_y, h_y, sizeof(real), hipMemcpyHostToDevice));
        reduce<true><<<grid_size, block_size, smem>>>(d_x, d_y, N);
    }
    else
    {
        reduce<false><<<grid_size, block_size, smem>>>(d_x, d_y, N);
    }

    if (!atomic && grid_size > 1)
    {
        reduce<false><<<1, block_size, smem>>>(d_y, d_y, grid_size);
    }

    CHECK(hipMemcpy(h_y, d_y, sizeof(real), hipMemcpyDeviceToHost));
    CHECK(hipFree(d_y));

    return h_y[0];
}

template<bool using_atomic>
void __global__ reduce(real *d_x, real *d_y, const int N)
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    extern __shared__ real s_y[];

    real y = 0.0;
    const int stride = blockDim.x * gridDim.x;
    for (int n = bid * blockDim.x + tid; n < N; n += stride)
    {
        y += d_x[n];
    }
    s_y[tid] = y;
    __syncthreads();

    for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1)
    {
        if (tid < offset)
        {
            s_y[tid] += s_y[tid + offset];
        }
        __syncthreads();
    }

    if (tid == 0)
    {
        if (using_atomic)
        {
            atomicAdd(d_y, s_y[0]);
        }
        else
        {
            d_y[bid] = s_y[0];
        }
    }
}



