#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <iostream>

int main(void)
{
    int N = 10;
    int *x, *y;
    hipMalloc((void **)&x, sizeof(int) * N);
    hipMalloc((void **)&y, sizeof(int) * N);
    int *h_x = (int*) malloc(sizeof(int) * N);
    for (int i = 0; i < N; ++i)
    {
        h_x[i] = i + 1;
    }
    hipMemcpy(x, h_x, sizeof(int) * N, hipMemcpyHostToDevice);

    thrust::device_ptr<int> x_ptr(x);
    thrust::device_ptr<int> y_ptr(y);
    thrust::inclusive_scan(x_ptr, x_ptr + N, y_ptr);
    y = thrust::raw_pointer_cast(y_ptr);

    int *h_y = (int*) malloc(sizeof(int) * N);
    hipMemcpy(h_y, y, sizeof(int) * N, hipMemcpyDeviceToHost);
    for (int i = 0; i < N; ++i)
    {
        std::cout << h_y[i] << std::endl;
    }

    hipFree(x);
    hipFree(y);
    free(h_x);
    free(h_y);
    return 0;
}

