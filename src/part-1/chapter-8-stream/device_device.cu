#include "hip/hip_runtime.h"
#include "error.cuh"
#include <math.h> // fabs()
#include <stdio.h>
#define EPSILON 1.0e-14 // a small number
void __global__ sum(double *x, double *y, double *z, int N);
void check(double *z, int N);
void run(int N_streams);

int main(void)
{
    for (int n = 0; n < 30; ++n)
    {
        run(n+1);
    }
    return 0;
}

void run(int N_streams)
{
    int N1 = 1000;
    int M1 = sizeof(double) * N1;
    int N_all = N1 * N_streams;
    int M_all = M1 * N_streams;
    double *x = (double*) malloc(M_all);
    double *y = (double*) malloc(M_all);
    double *z = (double*) malloc(M_all);
    for (int n = 0; n < N_all; ++n)
    {
        x[n] = 1.0;
        y[n] = 2.0;
        z[n] = 0.0;
    }
    double *g_x, *g_y, *g_z;
    CHECK(hipMalloc((void **)&g_x, M_all))
    CHECK(hipMalloc((void **)&g_y, M_all))
    CHECK(hipMalloc((void **)&g_z, M_all))
    CHECK(hipMemcpy(g_x, x, M_all, hipMemcpyHostToDevice))
    CHECK(hipMemcpy(g_y, y, M_all, hipMemcpyHostToDevice))

    hipStream_t *streams = (hipStream_t *) 
        malloc(N_streams * sizeof(hipStream_t));
    for (int i = 0 ; i < N_streams ; i++)
    {
        CHECK(hipStreamCreate(&(streams[i])));
    }

    hipEvent_t start;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventRecord(start, 0));

    for (int i = 0; i < N_streams; i++)
    {
        int offset = i * N1;
        int block_size = 128;
        int grid_size = (N1 - 1) / block_size + 1;
        sum<<<grid_size, block_size, 0, streams[i]>>>
        (g_x + offset, g_y + offset, g_z + offset, N1);
    }

    hipEvent_t stop;
    CHECK(hipEventCreate(&stop));
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    float elapsed_time;
    CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
    printf("Time with %d streams = %g ms\n", 
        N_streams, elapsed_time);

    for (int i = 0 ; i < N_streams; i++)
    {
        CHECK(hipStreamDestroy(streams[i]));
    }
    free(streams);
    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop));

    CHECK(hipMemcpy(z, g_z, M_all, hipMemcpyDeviceToHost))
    check(z, N_all);

    free(x);
    free(y);
    free(z);
    CHECK(hipFree(g_x))
    CHECK(hipFree(g_y))
    CHECK(hipFree(g_z))
}

void __global__ sum(double *x, double *y, double *z, int N)
{
    int n = blockDim.x * blockIdx.x + threadIdx.x;
    if (n < N)
    {
        for (int i = 0; i < 1000000; ++i)
        {
            z[n] = x[n] + y[n];
        }
    }
}

void check(double *z, int N)
{
    int has_error = 0;
    for (int n = 0; n < N; ++n)
    {
        has_error += (fabs(z[n] - 3.0) > EPSILON);
    }
    printf("%s\n", has_error ? "Has errors" : "No errors");
}

