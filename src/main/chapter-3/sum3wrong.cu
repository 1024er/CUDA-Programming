
#include <hip/hip_runtime.h>
#include <math.h> // fabs()
#include <stdio.h>
#define EPSILON 1.0e-14 // a small number
void __global__ sum(double *x, double *y, double *z, int N);
void check(double *z, int N);

int main(void)
{
    int N = 1024 * 100000;
    int M = sizeof(double) * N;
    // allocate host memory
    double *x = (double*) malloc(M);
    double *y = (double*) malloc(M);
    double *z = (double*) malloc(M);
    // initialize host data
    for (int n = 0; n < N; ++n)
    {
        x[n] = 1.0;
        y[n] = 2.0;
        z[n] = 0.0;
    }
    // allocate device memory
    double *g_x, *g_y, *g_z;
    hipMalloc((void **)&g_x, M);
    hipMalloc((void **)&g_y, M);
    hipMalloc((void **)&g_z, M);
    // copy data from host to device
    hipMemcpy(g_x, x, M, hipMemcpyHostToDevice);
    hipMemcpy(g_y, y, M, hipMemcpyHostToDevice);
    // call the kernel function
    int block_size = 128;
    int grid_size = N / block_size;
    sum<<<grid_size, block_size>>>(g_x, g_y, g_z, N);
    // copy data from device to host
    hipMemcpy(z, g_z, M, hipMemcpyDeviceToHost);
    // check the results
    check(z, N);
    // free host memory
    free(x);
    free(y);
    free(z);
    // gives segmentation fault
    free(g_x);
    free(g_y);
    free(g_z);
    return 0;
}

void __global__ sum(double *x, double *y, double *z, int N)
{
    int n = blockDim.x * blockIdx.x + threadIdx.x;
    z[n] = x[n] + y[n];
}

void check(double *z, int N)
{
    int has_error = 0;
    for (int n = 0; n < N; ++n)
    {
        has_error += (fabs(z[n] - 3.0) > EPSILON);
    }
    printf("%s\n", has_error ? "Has errors" : "No errors");
}

