
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
double get_length(double *x, int N);

int main(void)
{
    int N = 100000000;
    int M = sizeof(double) * N;
    double *x = (double *) malloc(M);
    for (int n = 0; n < N; ++n)
    {
        x[n] = 1.0;
    }
    double length = get_length(x, N);
    printf("length = %g.\n", length);
    free(x);
    return 0;
}

void __global__ get_length_1
(double *g_x, double *g_inner, int N)
{
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int n = bid * blockDim.x + tid;
    __shared__ double s_inner[128];
    s_inner[tid] = 0.0;

    if (n < N) 
    {
        double x_n = g_x[n];
        s_inner[tid] += x_n * x_n;
    }
    __syncthreads();

    for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1)
    {
        if (tid < offset)
        {
            s_inner[tid] += s_inner[tid + offset];
        }
        __syncthreads();
    }

    if (tid == 0)
    {
        g_inner[bid] = s_inner[0];
    }
}

void __global__ get_length_2
(double *g_inner, double *g_length, int N)
{
    int tid = threadIdx.x;
    __shared__ double s_length[1024];
    s_length[tid] = 0.0;

    int number_of_patches = (N - 1) / 1024 + 1; 
    for (int patch = 0; patch < number_of_patches; ++patch)
    {
        int n = tid + patch * 1024;
        if (n < N)
        {
            s_length[tid] += g_inner[n];
        }
    }
    __syncthreads();

    for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1)
    {
        if (tid < offset)
        {
            s_length[tid] += s_length[tid + offset];
        }
        __syncthreads();
    }

    if (tid == 0)
    {
        g_length[0] = s_length[0];
    }
}

double get_length(double *x, int N)
{
    int block_size = 128;
    int grid_size = (N - 1) / block_size + 1;
    double *g_inner;
    hipMalloc((void**)&g_inner, sizeof(double) * grid_size);
    double *g_length;
    hipMalloc((void**)&g_length, sizeof(double));
    double *g_x;
    hipMalloc((void**)&g_x, sizeof(double) * N);
    hipMemcpy(g_x, x, sizeof(double) * N, 
        hipMemcpyHostToDevice);

    get_length_1<<<grid_size, block_size>>>(g_x, g_inner, N);
    get_length_2<<<1, 1024>>>(g_inner, g_length, grid_size);

    double *cpu_length = (double *) malloc(sizeof(double));
    hipMemcpy(cpu_length, g_length, sizeof(double), 
        hipMemcpyDeviceToHost);
    hipFree(g_inner);
    hipFree(g_length);
    hipFree(g_x);
    double length = sqrt(cpu_length[0]);
    free(cpu_length);
    return length;
}



