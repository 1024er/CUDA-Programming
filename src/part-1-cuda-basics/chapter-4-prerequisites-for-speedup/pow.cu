#include "hip/hip_runtime.h"
#include "error.cuh"
#include <math.h> // fabs()
#include <stdio.h>
#include <time.h> // clock(), clock_t, and CLOCKS_PER_SEC
#define EPSILON 1.0e-14 // a small number
void __global__ power(double *x, double *y, double *z, int N);
void check(double *z, int N);

int main(void)
{
    int N = 100000000;
    int M = sizeof(double) * N;
    double *x = (double*) malloc(M);
    double *y = (double*) malloc(M);
    double *z = (double*) malloc(M);
    for (int n = 0; n < N; ++n) { x[n] = 1.0; y[n] = 2.0; }
    double *g_x, *g_y, *g_z;
    CHECK(hipMalloc((void **)&g_x, M))
    CHECK(hipMalloc((void **)&g_y, M))
    CHECK(hipMalloc((void **)&g_z, M))
    CHECK(hipMemcpy(g_x, x, M, hipMemcpyHostToDevice))
    CHECK(hipMemcpy(g_y, y, M, hipMemcpyHostToDevice))

    int block_size = 128;
    int grid_size = (N - 1) / block_size + 1;
    power<<<grid_size, block_size>>>(g_x, g_y, g_z, N);
   
    hipMemcpy(z, g_z, M, hipMemcpyDeviceToHost);
    check(z, N);
    free(x);
    free(y);
    free(z);
    CHECK(hipFree(g_x))
    CHECK(hipFree(g_y))
    CHECK(hipFree(g_z))
    return 0;
}

void __global__ power(double *x, double *y, double *z, int N)
{
    int n = blockDim.x * blockIdx.x + threadIdx.x;
    if (n < N)
    {
        z[n] = pow(x[n], y[n]);
    }
}

void check(double *z, int N)
{
    int has_error = 0;
    for (int n = 0; n < N; ++n)
    {
        has_error += (fabs(z[n] - 1.0) > EPSILON);
    }
    printf("%s\n", has_error ? "Has errors" : "No errors");
}

