#include "hip/hip_runtime.h"
#include "error.cuh"
void __global__ copy(double *x, double *y, int N);

int main(void)
{
    int N = 1 << 27;
    int M = sizeof(double) * N;
    // allocate host memory
    double *x = (double*) malloc(M);
    // initialize host data
    for (int n = 0; n < N; ++n)
    {
        x[n] = 1.0;
    }
    // allocate device memory
    double *g_x, *g_y;
    CHECK(hipMalloc((void **)&g_x, M))
    CHECK(hipMalloc((void **)&g_y, M))
    // copy data from host to device
    CHECK(hipMemcpy(g_x, x, M, hipMemcpyHostToDevice))
    // call the kernel function
    int block_size = 128;
    int grid_size = (N - 1) / block_size + 1;
    copy<<<grid_size, block_size>>>(g_x, g_y, N);
    // free host memory
    free(x);
    // free device memory
    CHECK(hipFree(g_x))
    CHECK(hipFree(g_y))
    return 0;
}

void __global__ copy(double *x, double *y, int N)
{
    int n = blockDim.x * blockIdx.x + threadIdx.x;
    y[n] = x[n];
}

