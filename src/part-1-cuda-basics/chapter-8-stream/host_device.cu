#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include "error.cuh"

void cpu_sum(double *x, double *y, double *z, int N);
void __global__ sum(double *x, double *y, double *z, int N);
void run(bool overlap);

int main(void)
{
    printf("Pre-computing on the GPU,\n");
    run(false);
    printf("Without CPU-GPU overlap,\n");
    run(false);
    printf("With CPU-GPU overlap,\n");
    run(true);
    return 0;
}

void cpu_sum(double *x, double *y, double *z, int N)
{
    for (int n = 0; n < N; ++n)
    {
        z[n] = x[n] + y[n];
    }
}

void __global__ sum(double *x, double *y, double *z, int N)
{
    int n = blockDim.x * blockIdx.x + threadIdx.x;
    if (n < N)
    {
        z[n] = x[n] + y[n];
    }
}

void run(bool overlap)
{
    int N = 100000000;
    int M = sizeof(double) * N;
    double *x = (double*) malloc(M);
    double *y = (double*) malloc(M);
    double *z = (double*) malloc(M);
    for (int n = 0; n < N; ++n)
    {
        x[n] = 1.0;
        y[n] = 2.0;
        z[n] = 0.0;
    }
    double *g_x, *g_y, *g_z;
    CHECK(hipMalloc((void **)&g_x, M))
    CHECK(hipMalloc((void **)&g_y, M))
    CHECK(hipMalloc((void **)&g_z, M))
    CHECK(hipMemcpy(g_x, x, M, hipMemcpyHostToDevice))
    CHECK(hipMemcpy(g_y, y, M, hipMemcpyHostToDevice))

    int block_size = 128;
    int grid_size = (N - 1) / block_size + 1;

    hipDeviceSynchronize();
    clock_t time_begin = clock();

    if (!overlap)
    {
        cpu_sum(x, y, z, N/25);
    }
    sum<<<grid_size, block_size>>>(g_x, g_y, g_z, N);
    if (overlap)
    {
        cpu_sum(x, y, z, N/25);
    }

    hipDeviceSynchronize();
    clock_t time_finish = clock();
    double time_used = (time_finish - time_begin)
        / double(CLOCKS_PER_SEC);
    printf("time used is %f s.\n", time_used);

    CHECK(hipMemcpy(z, g_z, M, hipMemcpyDeviceToHost))
    free(x);
    free(y);
    free(z);
    CHECK(hipFree(g_x))
    CHECK(hipFree(g_y))
    CHECK(hipFree(g_z))
}

