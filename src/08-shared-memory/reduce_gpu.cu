#include "hip/hip_runtime.h"
#include "error.cuh"
#include <stdio.h>

#ifdef USE_DP
    typedef double real;
#else
    typedef float real;
#endif

const int NUM_REPEATS = 10;

void timing(real *h_x, real *d_x, const int N, const int method);
real reduce(real *d_x, const int N, const int method);
void __global__ reduce_global(real *d_x, real *d_y, const int N);
void __global__ reduce_shared(real *d_x, real *d_y, const int N);
void __global__ reduce_dynamic(real *d_x, real *d_y, const int N);
void __global__ reduce_more(real *d_x, real *d_y, const int N);

int main(void)
{
    const int N = 100000000;
    const int M = sizeof(real) * N;
    real *h_x = (real *) malloc(M);
    for (int n = 0; n < N; ++n)
    {
        h_x[n] = 1.23;
    }
    real *d_x;
    CHECK(hipMalloc(&d_x, M));
;
    printf("\nusing global memory only:\n");
    timing(h_x, d_x, N, 0);
    printf("\nusing static shared memory:\n");
    timing(h_x, d_x, N, 1);
    printf("\nusing dynamic shared memory:\n");
    timing(h_x, d_x, N, 2);
    printf("\nusing dynamic shared memory:\n");
    timing(h_x, d_x, N, 3);

    free(h_x);
    CHECK(hipFree(d_x));
    return 0;
}

void timing(real *h_x, real *d_x, const int N, const int method)
{
    real sum = 0;
    float t_sum = 0;
    float t2_sum = 0;

    for (int repeat = 0; repeat <= NUM_REPEATS; ++repeat)
    {
        const int M = sizeof(real) * N;
        CHECK(hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice));

        hipEvent_t start, stop;
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        CHECK(hipEventRecord(start));

        sum = reduce(d_x, N, method);

        CHECK(hipEventRecord(stop));
        CHECK(hipEventSynchronize(stop));
        float elapsed_time;
        CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
        printf("Time = %g ms.\n", elapsed_time);

        if (repeat > 0)
        {
            t_sum += elapsed_time;
            t2_sum += elapsed_time * elapsed_time;
        }

        CHECK(hipEventDestroy(start));
        CHECK(hipEventDestroy(stop));
    }

    const float t_ave = t_sum / NUM_REPEATS;
    const float t_err = sqrt(t2_sum / NUM_REPEATS - t_ave * t_ave);
    printf("Time = %g +- %g ms.\n", t_ave, t_err);

    printf("sum = %f.\n", sum);
}

real reduce(real *d_x, const int N, const int method)
{
    const int block_size = 128;
    const int grid_size = (N + block_size - 1) / block_size;
    const int M = sizeof(real) * grid_size;
    const int smem = sizeof(real) * block_size;
    real *d_y;
    CHECK(hipMalloc(&d_y, M));
    real *h_y = (real *) malloc(M);

    switch (method)
    {
        case 0:
            reduce_global<<<grid_size, block_size>>>(d_x, d_y, N);
            break;
        case 1:
            reduce_shared<<<grid_size, block_size>>>(d_x, d_y, N);
            break;
        case 2:
            reduce_dynamic<<<grid_size, block_size, smem>>>(d_x, d_y, N);
            break;
        case 3:
            reduce_more<<<10000, block_size, smem>>>(d_x, d_y, N);
            break;
        default:
            printf("Error: wrong method\n");
            exit(1);
            break;
    }

    CHECK(hipMemcpy(h_y, d_y, M, hipMemcpyDeviceToHost));

    real result = 0.0;
    for (int n = 0; n < grid_size; ++n)
    {
        result += h_y[n];
    }

    free(h_y);
    CHECK(hipFree(d_y));
    return result;
}

void __global__ reduce_global(real *d_x, real *d_y, const int N)
{
    const int tid = threadIdx.x;
    real *x = d_x + blockDim.x * blockIdx.x;
    __syncthreads();

    for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1)
    {
        if (tid < offset)
        {
            x[tid] += x[tid + offset];
        }
        __syncthreads();
    }

    if (tid == 0)
    {
        d_y[blockIdx.x] = x[0];
    }
}

void __global__ reduce_shared(real *d_x, real *d_y, const int N)
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int n = bid * blockDim.x + tid;
    __shared__ real s_y[128];
    s_y[tid] = (n < N) ? d_x[n] : 0.0;
    __syncthreads();

    for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1)
    {

        if (tid < offset)
        {
            s_y[tid] += s_y[tid + offset];
        }
        __syncthreads();
    }

    if (tid == 0)
    {
        d_y[bid] = s_y[0];
    }
}


void __global__ reduce_dynamic(real *d_x, real *d_y, const int N)
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int n = bid * blockDim.x + tid;
    extern __shared__ real s_y[];
    s_y[tid] = (n < N) ? d_x[n] : 0.0;
    __syncthreads();

    for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1)
    {

        if (tid < offset)
        {
            s_y[tid] += s_y[tid + offset];
        }
        __syncthreads();
    }

    if (tid == 0)
    {
        d_y[bid] = s_y[0];
    }
}

void __global__ reduce_more(real *d_x, real *d_y, const int N)
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    extern __shared__ real s_y[];

    real y = 0.0;
    for (int n = bid * blockDim.x + tid; n < N; n += blockDim.x * gridDim.x)
    {
        y += d_x[n];
    }
    s_y[tid] = y;
    __syncthreads();

    for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1)
    {
        if (tid < offset)
        {
            s_y[tid] += s_y[tid + offset];
        }
        __syncthreads();
    }

    if (tid == 0)
    {
        d_y[bid] = s_y[0];
    }
}


