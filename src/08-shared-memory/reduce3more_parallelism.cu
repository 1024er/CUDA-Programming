#include "hip/hip_runtime.h"
#include "error.cuh"
#include <stdio.h>

#ifdef USE_DP
    typedef double real;
#else
    typedef float real;
#endif

const int NUM_ROUNDS = 10;

real reduce(real *x, int N);

int main(int argc, char **argv)
{
    int N = 100000000;
    int M = sizeof(real) * N;
    real *h_x = (real *)malloc(M);
    for (int n = 0; n < N; ++n) { h_x[n] = 1.0; }
    real *x;
    CHECK(hipMalloc(&x, M))
    CHECK(hipMemcpy(x, h_x, M, hipMemcpyHostToDevice))

    real sum = reduce(x, N);
    printf("sum = %f.\n", sum);

    free(h_x);
    CHECK(hipFree(x))
    return 0;
}

void __global__ reduce(real *g_x, real *g_y, int N)
{
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    __shared__ real s_y[128];

    real y = 0.0;
    int offset = tid + bid * blockDim.x * NUM_ROUNDS;
    for (int round = 0; round < NUM_ROUNDS; ++round)
    {
        int n = round * blockDim.x + offset;
        if (n < N) { y += g_x[n]; }
    }
    s_y[tid] = y;

    for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1)
    {
        __syncthreads();
        if (tid < offset) { s_y[tid] += s_y[tid + offset]; }
    }

    if (tid == 0) { g_y[bid] = s_y[0]; }
}

real reduce(real *x, int N)
{
    const int block_size = 128;
    int grid_size = (N - 1) / (block_size * NUM_ROUNDS) + 1;

    real *y;
    CHECK(hipMalloc(&y, sizeof(real) * grid_size))

    reduce<<<grid_size, block_size>>>(x, y, N);

    real *h_y = (real *)malloc(sizeof(real) * grid_size);
    CHECK(hipMemcpy(h_y, y, sizeof(real) * grid_size, 
        hipMemcpyDeviceToHost))

    real result = 0.0;
    for (int n = 0; n < grid_size; ++n) { result += h_y[n]; }

    free(h_y);
    CHECK(hipFree(y))
    return result;
}

