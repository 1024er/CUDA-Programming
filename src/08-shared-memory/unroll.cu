#include "hip/hip_runtime.h"
#include "error.cuh"
#include <stdio.h>

#ifdef USE_DP
    typedef double real;
#else
    typedef float real;
#endif

const int TILE_DIM = 32;
const int BLOCK_Y = 4;

__global__ void transpose(real *A, real *B, int N);
void print_matrix(int N, real *A);

int main(int argc, char **argv)
{
    int N = atoi(argv[1]);
    int N2 = N * N;
    int grid_size_x = (N - 1) / TILE_DIM + 1;
    int grid_size_y = (N - 1) / TILE_DIM + 1;
    dim3 block_size(TILE_DIM, BLOCK_Y);
    dim3 grid_size(grid_size_x, grid_size_y);

    int M = sizeof(real) * N2;
    real *h_A = (real *)malloc(M);
    real *h_B = (real *)malloc(M);
    for (int n = 0; n < N2; ++n) { h_A[n] = n; }
    real *A, *B;
    CHECK(hipMalloc(&A, M))
    CHECK(hipMalloc(&B, M))
    CHECK(hipMemcpy(A, h_A, M, hipMemcpyHostToDevice))

    transpose<<<grid_size, block_size>>>(A, B, N);

    CHECK(hipMemcpy(h_B, B, M, hipMemcpyDeviceToHost))
    if (N <= 10)
    {
        printf("A =\n");
        print_matrix(N, h_A);
        printf("\nB = transpose(A) =\n");
        print_matrix(N, h_B);
    }

    free(h_A); free(h_B);
    CHECK(hipFree(A))
    CHECK(hipFree(B))
    return 0;
}

__global__ void transpose(real *A, real *B, int N)
{
    __shared__ real S[TILE_DIM][TILE_DIM + 1];
    int bx = blockIdx.x * TILE_DIM;
    int by = blockIdx.y * TILE_DIM;

    int nx1 = bx + threadIdx.x;
    int ny1 = by + threadIdx.y;
    for (int y = 0; y < TILE_DIM; y += BLOCK_Y)
    {
        if (nx1 < N && (ny1 + y) < N)
        {
            S[threadIdx.y + y][threadIdx.x] = A[(ny1 + y) * N + nx1];
        }
    }
    __syncthreads();

    int nx2 = bx + threadIdx.y;
    int ny2 = by + threadIdx.x;
    for (int y = 0; y < TILE_DIM; y += BLOCK_Y)
    {
        if ((nx2 + y) < N && ny2 < N)
        {
            B[(nx2 + y) * N + ny2] = S[threadIdx.x][threadIdx.y + y];
        }
    }
}

void print_matrix(int N, real *A)
{
    for (int ny = 0; ny < N; ny++)
    {
        for (int nx = 0; nx < N; nx++)
        {
            printf("%g\t", A[ny * N + nx]);
        }
        printf("\n");
    }
}

