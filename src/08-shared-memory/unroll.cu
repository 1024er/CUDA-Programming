#include "hip/hip_runtime.h"
#include "error.cuh"
#include <stdio.h>

#ifdef USE_DP
    typedef double real;
#else
    typedef float real;
#endif

const int TILE_DIM = 32;
const int BLOCK_Y = 4;

__global__ void transpose(real *A, real *B, int N);
void print_matrix(int N, real *A);

int main(int argc, char **argv)
{
    int N = atoi(argv[1]);
    int N2 = N * N;
    int grid_size_x = (N - 1) / TILE_DIM + 1;
    int grid_size_y = (N - 1) / TILE_DIM + 1;
    dim3 block_size(TILE_DIM, BLOCK_Y);
    dim3 grid_size(grid_size_x, grid_size_y);

    int M = sizeof(real) * N2;
    real *h_A = (real *)malloc(M);
    real *h_B = (real *)malloc(M);
    for (int n = 0; n < N2; ++n) { h_A[n] = n; }
    real *A, *B;
    CHECK(hipMalloc(&A, M))
    CHECK(hipMalloc(&B, M))
    CHECK(hipMemcpy(A, h_A, M, hipMemcpyHostToDevice))

    transpose<<<grid_size, block_size>>>(A, B, N);

    CHECK(hipMemcpy(h_B, B, M, hipMemcpyDeviceToHost))
    if (N <= 10)
    {
        printf("A =\n");
        print_matrix(N, h_A);
        printf("\nB = transpose(A) =\n");
        print_matrix(N, h_B);
    }

    free(h_A); free(h_B);
    CHECK(hipFree(A))
    CHECK(hipFree(B))
    return 0;
}

__global__ void transpose(real *A, real *B, int N)
{
    __shared__ real S[TILE_DIM][TILE_DIM + 1];

    int nx = blockIdx.x * TILE_DIM + threadIdx.x;
    int ny = blockIdx.y * TILE_DIM + threadIdx.y;
    for (int y = 0; y < TILE_DIM; y += BLOCK_Y)
    {
        if (nx < N && (ny + y) < N)
        {
            S[threadIdx.y + y][threadIdx.x] = A[(ny + y) * N + nx];
        }
    }
    __syncthreads();

    nx = blockIdx.x * TILE_DIM + threadIdx.y;
    ny = blockIdx.y * TILE_DIM + threadIdx.x;
    for (int y = 0; y < TILE_DIM; y += BLOCK_Y)
    {
        if ((nx + y) < N && ny < N)
        {
            B[(nx + y) * N + ny] = S[threadIdx.x][threadIdx.y + y];
        }
    }
}

void print_matrix(int N, real *A)
{
    for (int ny = 0; ny < N; ny++)
    {
        for (int nx = 0; nx < N; nx++)
        {
            printf("%g\t", A[ny * N + nx]);
        }
        printf("\n");
    }
}

