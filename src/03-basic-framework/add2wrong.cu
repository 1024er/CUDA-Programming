
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#define EPSILON 1.0e-14
void __global__ add(double *x, double *y, double *z, int N);
void check(double *z, int N);

int main(void)
{
    int N = 100000000;
    int M = sizeof(double) * N;
    double *x = (double*) malloc(M);
    double *y = (double*) malloc(M);
    double *z = (double*) malloc(M);
    for (int n = 0; n < N; ++n)
    {
        x[n] = 1.0; y[n] = 2.0; z[n] = 0.0;
    }

    double *g_x, *g_y, *g_z;
    hipMalloc((void **)&g_x, M);
    hipMalloc((void **)&g_y, M);
    hipMalloc((void **)&g_z, M);
    hipMemcpy(g_x, x, M, hipMemcpyDeviceToHost);
    hipMemcpy(g_y, y, M, hipMemcpyDeviceToHost);

    int block_size = 128;
    int grid_size = N / block_size;
    add<<<grid_size, block_size>>>(g_x, g_y, g_z, N);

    hipMemcpy(z, g_z, M, hipMemcpyDeviceToHost);
    check(z, N);

    free(x); free(y); free(z);
    hipFree(g_x); hipFree(g_y); hipFree(g_z);
    return 0;
}

void __global__ add(double *x, double *y, double *z, int N)
{
    int n = blockDim.x * blockIdx.x + threadIdx.x;
    z[n] = x[n] + y[n];
}

void check(double *z, int N)
{
    int has_error = 0;
    for (int n = 0; n < N; ++n)
    {
        has_error += (fabs(z[n] - 3.0) > EPSILON);
    }
    printf("%s\n", has_error ? "Has errors" : "No errors");
}

