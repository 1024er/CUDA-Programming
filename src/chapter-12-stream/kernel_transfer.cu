#include "hip/hip_runtime.h"
#include "error.cuh"
#include <stdio.h>
#include <math.h>
#define EPSILON 1.0e-14

void __global__ sum(double *x, double *y, double *z, int N);
void run(int N_streams);

void check(double *z, int N)
{
    int has_error = 0;
    for (int n = 0; n < N; ++n)
    {
        has_error += (fabs(z[n] - 3.0) > EPSILON);
    }
    printf("%s\n", has_error ? "Has errors" : "No errors");
}

int main(void)
{
    for (int n = 1; n < 128; n *=2)
    {
        run(n);
    }
    return 0;
}

void run(int N_streams)
{
    int N_all = 1 << 20;
    int M_all = sizeof(double) * N_all;
    int N1 = N_all / N_streams;
    int M1 = M_all / N_streams;

    double *x, *y, *z;
    CHECK(hipHostMalloc((void**)&x, M_all));
    CHECK(hipHostMalloc((void**)&y, M_all));
    CHECK(hipHostMalloc((void**)&z, M_all));
    for (int n = 0; n < N_all; ++n)
    {
        x[n] = 1.0;
        y[n] = 2.0;
        z[n] = 0.0;
    }
    double *g_x, *g_y, *g_z;
    CHECK(hipMalloc((void **)&g_x, M_all))
    CHECK(hipMalloc((void **)&g_y, M_all))
    CHECK(hipMalloc((void **)&g_z, M_all))

    hipStream_t *streams = (hipStream_t *) 
        malloc(N_streams * sizeof(hipStream_t));
    for (int i = 0; i < N_streams; i++)
    {
        CHECK(hipStreamCreate(&(streams[i])));
    }

    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));
    CHECK(hipEventRecord(start));

    for (int i = 0; i < N_streams; i++)
    {
        int offset = i * N1;
        CHECK(hipMemcpyAsync(g_x + offset, x + offset, M1, 
            hipMemcpyHostToDevice, streams[i]));
        CHECK(hipMemcpyAsync(g_y + offset, y + offset, M1, 
            hipMemcpyHostToDevice, streams[i]));

        int block_size = 128;
        int grid_size = (N1 - 1) / block_size + 1;
        sum<<<grid_size, block_size, 0, streams[i]>>>
        (g_x + offset, g_y + offset, g_z + offset, N1);

        CHECK(hipMemcpyAsync(z + offset, g_z + offset, M1, 
            hipMemcpyDeviceToHost, streams[i]));
    }

    CHECK(hipEventRecord(stop));
    CHECK(hipEventSynchronize(stop));
    float elapsed_time;
    CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
    printf("%d\t%g\n", N_streams, elapsed_time);

    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop));

    for (int i = 0 ; i < N_streams; i++)
    {
        CHECK(hipStreamDestroy(streams[i]));
    }
    free(streams);

    hipDeviceSynchronize();
    check(z, N_all);

    CHECK(hipHostFree(x))
    CHECK(hipHostFree(y))
    CHECK(hipHostFree(z))
    CHECK(hipFree(g_x))
    CHECK(hipFree(g_y))
    CHECK(hipFree(g_z))
}

void __global__ sum(double *x, double *y, double *z, int N)
{
    int n = blockDim.x * blockIdx.x + threadIdx.x;
    if (n < N)
    {
        for (int i = 0; i < 40; ++i)
        {
            z[n] = x[n] + y[n];
        }
    }
}
