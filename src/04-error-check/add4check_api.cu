#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include "error.cuh"

const double EPSILON = 1.0e-14;
void __global__ add(const double *x, const double *y, double *z, const int N);
void check(const double *z, const int N);

int main(void)
{
    const int N = 100000000;
    const int M = sizeof(double) * N;
    double *x = (double*) malloc(M);
    double *y = (double*) malloc(M);
    double *z = (double*) malloc(M);
    for (int n = 0; n < N; ++n)
    {
        x[n] = 1.0;
        y[n] = 2.0;
    }

    double *g_x, *g_y, *g_z;
    CHECK(hipMalloc((void **)&g_x, M))
    CHECK(hipMalloc((void **)&g_y, M))
    CHECK(hipMalloc((void **)&g_z, M))
    CHECK(hipMemcpy(g_x, x, M, hipMemcpyDeviceToHost))
    CHECK(hipMemcpy(g_y, y, M, hipMemcpyDeviceToHost))
    const int block_size = 128;
    const int grid_size = (N - 1) / block_size + 1;
    add<<<grid_size, block_size>>>(g_x, g_y, g_z, N);

    CHECK(hipMemcpy(z, g_z, M, hipMemcpyDeviceToHost))
    check(z, N);

    free(x);
    free(y);
    free(z);
    CHECK(hipFree(g_x))
    CHECK(hipFree(g_y))
    CHECK(hipFree(g_z))
    return 0;
}

void __global__ add(const double *x, const double *y, double *z, const int N)
{
    int n = blockDim.x * blockIdx.x + threadIdx.x;
    if (n < N)
    {
        z[n] = x[n] + y[n];
    }
}

void check(const double *z, const int N)
{
    bool has_error = false;
    for (int n = 0; n < N; ++n)
    {
        if (fabs(z[n] - 3.0) > EPSILON)
        {
            has_error = true;
        }
    }
    printf("%s\n", has_error ? "Has errors" : "No errors");
}

