#include "hip/hip_runtime.h"
#include "error.cuh"
#include <stdio.h>
#ifdef USE_DP
    typedef double real;
#else
    typedef float real;
#endif

real reduce(real *x, int N, int M);

int main(int argc, char **argv)
{
    int M = atoi(argv[1]);
    int N = 100000000;
    real *x;
    CHECK(hipMallocManaged(&x, sizeof(real) * N))
    for (int n = 0; n < N; ++n) { x[n] = 1.0; }

    real sum = reduce(x, N, M);
    printf("sum = %g.\n", sum);
    CHECK(hipFree(x))
    return 0;
}

void __global__ reduce_1
(real *g_x, real *g_y, int N, int number_of_rounds)
{
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    __shared__ real s_sum;
    s_sum = 0.0;
    real y = 0.0;
    int offset = tid + bid * blockDim.x * number_of_rounds;
    for (int round = 0; round < number_of_rounds; ++round)
    {
        int n = round * blockDim.x + offset;
        if (n < N) { y += g_x[n]; }
    }
    __syncthreads();

    atomicAdd(&s_sum, y);
    __syncthreads();
    if (tid == 0) { atomicAdd(g_y, s_sum); }
}

real reduce(real *x, int N, int M)
{
    int block_size = 128;
    int grid_size = (N - 1) / (block_size * M) + 1;

    real *sum;
    CHECK(hipMallocManaged(&sum, sizeof(real)))
    CHECK(hipDeviceSynchronize())
    sum[0] = 0.0;

    reduce_1<<<grid_size, block_size>>>(x, sum, N, M);

    CHECK(hipDeviceSynchronize())
    real result = sum[0];
    CHECK(hipFree(sum))
    return result;
}

