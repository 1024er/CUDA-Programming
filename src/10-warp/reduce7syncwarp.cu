#include "hip/hip_runtime.h"
#include "error.cuh"
#include <stdio.h>
#ifdef USE_DP
    typedef double real;
#else
    typedef float real;
#endif
real reduce(real *x, int N, int K);

int main(int argc, char **argv)
{
    int K = atoi(argv[1]);
    int N = 100000000;
    int M = sizeof(real) * N;
    real *h_x = (real *)malloc(M);
    for (int n = 0; n < N; ++n) { h_x[n] = 1.0; }
    real *x;
    CHECK(hipMalloc(&x, M))
    CHECK(hipMemcpy(x, h_x, M, hipMemcpyHostToDevice))

    real sum = reduce(x, N, K);
    printf("sum = %g.\n", sum);

    free(h_x);
    CHECK(hipFree(x))
    return 0;
}

void __global__ reduce_1
(real *g_x, real *g_y, int N, int number_of_rounds)
{
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    __shared__ real s_y[128];

    real y = 0.0;
    int offset = tid + bid * blockDim.x * number_of_rounds;
    for (int round = 0; round < number_of_rounds; ++round)
    {
        int n = round * blockDim.x + offset;
        if (n < N) { y += g_x[n]; }
    }
    s_y[tid] = y;
    __syncthreads();

    for (int offset = blockDim.x >> 1; offset > 32; offset >>= 1)
    {
        if (tid < offset) { s_y[tid] += s_y[tid + offset]; }
        __syncthreads();
    }
    for (int offset = 32; offset > 0; offset >>= 1)
    {
        if (tid < offset) { s_y[tid] += s_y[tid + offset]; }
        __syncwarp();
    }
    if (tid == 0) { atomicAdd(g_y, s_y[0]); }
}

real reduce(real *x, int N, int K)
{
    int block_size = 128;
    int grid_size = (N - 1) / (block_size * K) + 1;

    real *h_sum = (real *)malloc(sizeof(real));
    h_sum[0] = 0.0;
    real *sum;
    CHECK(hipMalloc(&sum, sizeof(real)))
    CHECK(hipMemcpy(sum, h_sum, sizeof(real), 
        hipMemcpyHostToDevice))

    reduce_1<<<grid_size, block_size>>>(x, sum, N, K);

    CHECK(hipMemcpy(h_sum, sum, sizeof(real), 
        hipMemcpyDeviceToHost))
    real result = h_sum[0];

    free(h_sum);
    CHECK(hipFree(sum))
    return result;
}

