
#include <hip/hip_runtime.h>
#include <stdlib.h> // malloc() and free()
#include <stdio.h> // printf()
#include <math.h> // sqrt()
#include <time.h> // clock_t, clock(), and CLOCKS_PER_SEC
double get_length(double *x, int N);

int main(void)
{
    int N = 1000;
    int M = sizeof(double) * N;
    double *x = (double *) malloc(M);
    for (int n = 0; n < N; ++n)
    {
        x[n] = 1.0;
    }
    double length = get_length(x, N);
    printf("length = %g.\n", length);
    free(x);
    return 0;
}

void __device__ warp_reduce(volatile double *s, int t) 
{
    s[t] += s[t + 32]; s[t] += s[t + 16]; s[t] += s[t + 8];
    s[t] += s[t + 4];  s[t] += s[t + 2];  s[t] += s[t + 1];
}

void __global__ get_length(double *g_x, double *g_length, int N)
{
    int tid = threadIdx.x;
    __shared__ double s_x[1024];
    s_x[tid] = 0.0;
    if (tid < N)
    {
        double tmp = g_x[tid];
        s_x[tid] = tmp * tmp;
    }
    __syncthreads();
    if (tid < 512) { s_x[tid] += s_x[tid + 512]; }
    __syncthreads();
    if (tid < 256) { s_x[tid] += s_x[tid + 256]; } 
    __syncthreads();
    if (tid < 128) { s_x[tid] += s_x[tid + 128]; } 
    __syncthreads();
    if (tid <  64) { s_x[tid] += s_x[tid + 64]; }  
    __syncthreads();
    if (tid < 32)
    { 
        warp_reduce(s_x, tid); 
    }
    if (tid == 0)
    {
        g_length[0] = sqrt(s_x[0]);
    }
}

double get_length(double *x, int N)
{
    double *g_length;
    hipMalloc((void**)&g_length, sizeof(double));
    double *g_x;
    hipMalloc((void**)&g_x, sizeof(double) * N);
    hipMemcpy(g_x, x, sizeof(double) * N, hipMemcpyHostToDevice);
    get_length<<<1, 1024>>>(g_x, g_length, N);
    double *cpu_length = (double *) malloc(sizeof(double));
    hipMemcpy(cpu_length, g_length, sizeof(double), hipMemcpyDeviceToHost);
    hipFree(g_length);
    hipFree(g_x);
    double length = cpu_length[0];
    free(cpu_length);
    return length;
}

