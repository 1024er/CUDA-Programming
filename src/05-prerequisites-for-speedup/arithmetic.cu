#include "hip/hip_runtime.h"
#include "error.cuh"
#include <math.h>
#include <stdio.h>
#include <time.h>
#define EPSILON 1.0e-14

#ifdef USE_DP
    typedef double real;
    #define EPSILON 1.0e-14
#else
    typedef float real;
    #define EPSILON 1.0e-6
#endif
void __global__ arithmetic(real *x, real *y, int N);
void check(real *z, int N);

int main(int argc, char **argv)
{
    int N = atoi(argv[1]);
    int block_size = atoi(argv[2]);
    int M = sizeof(real) * N;
    real *x = (real*) malloc(M);
    real *y = (real*) malloc(M);
    for (int n = 0; n < N; ++n)
    {
        x[n] = 2.0; y[n] = 0.0;
    }
    real *g_x, *g_y;
    CHECK(hipMalloc((void **)&g_x, M))
    CHECK(hipMalloc((void **)&g_y, M))
    CHECK(hipMemcpy(g_x, x, M, hipMemcpyHostToDevice))

    int grid_size = (N - 1) / block_size + 1;
    arithmetic<<<grid_size, block_size>>>(g_x, g_y, N);
   
    hipMemcpy(y, g_yy, M, hipMemcpyDeviceToHost);
    check(, N);

    free(x); free(y); free(z);
    CHECK(hipFree(g_x))
    CHECK(hipFree(g_y))
    CHECK(hipFree(g_z))
    return 0;
}

void __global__ arithmetic(real *x, real *y, int N)
{
    int n = blockDim.x * blockIdx.x + threadIdx.x;
    if (n < N) 
    {
        real x1 = x[n];
        real x30 = pow(x1, 30.0);
        real sin_x = sin(x30);
        real cos_x = cos(x30);
        y[n] = sin_x * sin_x + cos_x * cos_x;
    }
}

void check(real *y, int N)
{
    int has_error = 0;
    for (int n = 0; n < N; ++n)
    {
        has_error += (fabs(y[n] - 1.0) > EPSILON);
    }
    printf("%s\n", has_error ? "Has errors" : "No errors");
}

