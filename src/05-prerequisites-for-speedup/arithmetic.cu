#include "hip/hip_runtime.h"
#include "error.cuh"
#include <math.h>
#include <stdio.h>
#include <time.h>

#ifdef USE_DP
    typedef double real;
    #define EPSILON 1.0e-14
#else
    typedef float real;
    #define EPSILON 1.0e-6
#endif
void __global__ arithmetic(real *x, int N);
void check(real *x, int N);

int main(int argc, char **argv)
{
    int N = atoi(argv[1]);
    int block_size = atoi(argv[2]);
    int M = sizeof(real) * N;
    real *x = (real*) malloc(M);
    real *g_x;
    CHECK(hipMalloc((void **)&g_x, M))

    int grid_size = (N - 1) / block_size + 1;
    arithmetic<<<grid_size, block_size>>>(g_x, N);
   
    hipMemcpy(x, g_x, M, hipMemcpyDeviceToHost);
    check(x, N);

    free(x);
    CHECK(hipFree(g_x))

    return 0;
}

void __global__ arithmetic(real *g_x, int N)
{
    int n = blockDim.x * blockIdx.x + threadIdx.x;
    if (n < N) 
    {
        real a = 0;
        for (int m = 0; m < 1000; ++m)
        {
            a++;
        }
        g_x[n] = a;
    }
}

void check(real *x, int N)
{
    int has_error = 0;
    for (int n = 0; n < N; ++n)
    {
        has_error += (fabs(x[n] - 1000.0) > EPSILON);
    }
    printf("%s\n", has_error ? "Has errors" : "No errors");
}

