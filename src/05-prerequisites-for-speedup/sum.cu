#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include "error.cuh"

#ifdef USE_DP
    typedef double real;
    #define EPSILON 1.0e-14
#else
    typedef float real;
    #define EPSILON 1.0e-6
#endif
void __global__ sum(real *x, real *y, real *z, int N);
void check(real *z, int N);

int main(int argc, char **argv)
{
    if (argc != 1) 
    { printf("requires 1 argument\n"); exit(1); }
    int num_of_repeats = atoi(argv[1]);

    int N = 100000000;
    int M = sizeof(real) * N;
    real *x = (real*) malloc(M);
    real *y = (real*) malloc(M);
    real *z = (real*) malloc(M);
    for (int n = 0; n < N; ++n)
    {
        x[n] = 1.0; y[n] = 2.0; z[n] = 0.0;
    }
    real *g_x, *g_y, *g_z;
    CHECK(hipMalloc((void **)&g_x, M))
    CHECK(hipMalloc((void **)&g_y, M))
    CHECK(hipMalloc((void **)&g_z, M))
    CHECK(hipMemcpy(g_x, x, M, hipMemcpyHostToDevice))
    CHECK(hipMemcpy(g_y, y, M, hipMemcpyHostToDevice))

    int block_size = 128;
    int grid_size = (N - 1) / block_size + 1;
    for (int n = 0; n < num_of_repeats; ++n)
    {
        sum<<<grid_size, block_size>>>(g_x, g_y, g_z, N);
    }

    CHECK(hipMemcpy(z, g_z, M, hipMemcpyDeviceToHost))
    check(z, N);

    free(x); free(y); free(z);
    CHECK(hipFree(g_x))
    CHECK(hipFree(g_y))
    CHECK(hipFree(g_z))
    return 0;
}

void __global__ sum(real *x, real *y, real *z, int N)
{
    int n = blockDim.x * blockIdx.x + threadIdx.x;
    if (n < N) { z[n] = x[n] + y[n]; }
}

void check(real *z, int N)
{
    int has_error = 0;
    for (int n = 0; n < N; ++n)
    {
        has_error += (fabs(z[n] - 3.0) > EPSILON);
    }
    printf("%s\n", has_error ? "Has errors" : "No errors");
}

