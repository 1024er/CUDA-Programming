#include "hip/hip_runtime.h"
#include "error.cuh"
#include <stdio.h>
#ifdef USE_DP
    typedef double real;
#else
    typedef float real;
#endif
real reduce(real *x, int N);

int main(void)
{
    int N = 100000000;
    int M = sizeof(real) * N;
    real *x;
    CHECK(hipMallocManaged(&x, M))
    for (int n = 0; n < N; ++n) { x[n] = 1.0; }

    real sum = reduce(x, N);
    printf("sum = %g.\n", sum);
    CHECK(hipFree(x))
    return 0;
}

void __global__ reduce
(real *g_x, real *g_sum, int N, int number_of_rounds)
{
    int tid = threadIdx.x;
    __shared__ real s_sum[1024];
    real tmp_sum = 0.0;
    for (int round = 0; round < number_of_rounds; ++round)
    {
        int n = tid + round * 1024;
        if (n < N) { tmp_sum += g_x[n]; }
    }
    s_sum[tid] = tmp_sum;
    __syncthreads();

    for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1)
    {
        if (tid < offset) { s_sum[tid] += s_sum[tid + offset]; }
        __syncthreads();
    }
    
    if (tid == 0) { g_sum[0] = s_sum[0]; }
}

real reduce(real *x, int N)
{
    real *sum;
    CHECK(hipMallocManaged(&sum, sizeof(real)))
    
    int number_of_rounds = (N - 1) / 1024 + 1;
    reduce<<<1, 1024>>>(x, sum, N, number_of_rounds);

    CHECK(hipDeviceSynchronize())
    real result = sum[0];
    CHECK(hipFree(sum))
    return result;
}

