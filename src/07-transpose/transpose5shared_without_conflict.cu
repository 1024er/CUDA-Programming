#include "hip/hip_runtime.h"
#include "error.cuh"
#include <stdio.h>
#define TILE_DIM 32
#ifdef USE_DP
    typedef double real;
#else
    typedef float real;
#endif

__global__ void transpose(real *A, real *B, int N);
void print_matrix(int N, real *A);

int main(int argc, char **argv)
{
    int N = atoi(argv[1]);
    int N2 = N * N;
    int grid_size_x = (N - 1) / TILE_DIM + 1;
    int grid_size_y = (N - 1) / TILE_DIM + 1;
    dim3 block_size(TILE_DIM, TILE_DIM);
    dim3 grid_size(grid_size_x, grid_size_y);

    int M = sizeof(real) * N2;
    real *h_A = (real *)malloc(M);
    real *h_B = (real *)malloc(M);
    for (int n = 0; n < N2; ++n) { h_A[n] = n; }
    real *A, *B;
    CHECK(hipMalloc(&A, M))
    CHECK(hipMalloc(&B, M))
    CHECK(hipMemcpy(A, h_A, M, hipMemcpyHostToDevice))

    transpose<<<grid_size, block_size>>>(A, B, N);

    CHECK(hipMemcpy(h_B, B, M, hipMemcpyDeviceToHost))
    if (N <= 10)
    {
        printf("A =\n");
        print_matrix(N, h_A);
        printf("\nB = transpose(A) =\n");
        print_matrix(N, h_B);
    }

    free(h_A); free(h_B);
    CHECK(hipFree(A))
    CHECK(hipFree(B))
    return 0;
}

__global__ void transpose(real *A, real *B, int N)
{
    __shared__ real S[TILE_DIM][TILE_DIM + 1];
    int bx = blockIdx.x * blockDim.x;
    int by = blockIdx.y * blockDim.y;

    int nx1 = bx + threadIdx.x;
    int ny1 = by + threadIdx.y;
    if (nx1 < N && ny1 < N)
    {
        S[threadIdx.y][threadIdx.x] = A[ny1 * N + nx1];
    }
    __syncthreads();

    int nx2 = bx + threadIdx.y;
    int ny2 = by + threadIdx.x;
    if (nx2 < N && ny2 < N)
    {
        B[nx2 * N + ny2] = S[threadIdx.x][threadIdx.y];
    }
}

void print_matrix(int N, real *A)
{
    for (int ny = 0; ny < N; ny++)
    {
        for (int nx = 0; nx < N; nx++)
        {
            printf("%g\t", A[ny * N + nx]);
        }
        printf("\n");
    }
}

