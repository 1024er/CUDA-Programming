#include "error.cuh"
#include "memory.h"
#include <stdlib.h>

void allocate_memory(int N, int MN, Atom *atom)
{
    atom->m  = (double*) malloc(N * sizeof(double));
    atom->x  = (double*) malloc(N * sizeof(double));
    atom->y  = (double*) malloc(N * sizeof(double));
    atom->z  = (double*) malloc(N * sizeof(double));
    atom->vx = (double*) malloc(N * sizeof(double));
    atom->vy = (double*) malloc(N * sizeof(double));
    atom->vz = (double*) malloc(N * sizeof(double));
    atom->fx = (double*) malloc(N * sizeof(double));
    atom->fy = (double*) malloc(N * sizeof(double));
    atom->fz = (double*) malloc(N * sizeof(double));
    atom->pe = (double*) malloc(N * sizeof(double));
    atom->box = (double*) malloc(6 * sizeof(double));

    CHECK(hipMalloc((void**)&atom->g_NN, sizeof(int) * N))
    CHECK(hipMalloc((void**)&atom->g_NL, sizeof(int) * N * MN))
    CHECK(hipMalloc((void**)&atom->g_x, sizeof(double) * N))
    CHECK(hipMalloc((void**)&atom->g_y, sizeof(double) * N))
    CHECK(hipMalloc((void**)&atom->g_z, sizeof(double) * N))
    CHECK(hipMalloc((void**)&atom->g_fx, sizeof(double) * N))
    CHECK(hipMalloc((void**)&atom->g_fy, sizeof(double) * N))
    CHECK(hipMalloc((void**)&atom->g_fz, sizeof(double) * N))
    CHECK(hipMalloc((void**)&atom->g_pe, sizeof(double) * N))
    CHECK(hipMalloc((void**)&atom->g_box, sizeof(double) * 6))
}

void deallocate_memory(Atom *atom)
{
    free(atom->m);
    free(atom->x);
    free(atom->y);
    free(atom->z);
    free(atom->vx);
    free(atom->vy);
    free(atom->vz);
    free(atom->fx);
    free(atom->fy);
    free(atom->fz);
    free(atom->pe);
    free(atom->box);

    CHECK(hipFree(atom->g_NN)) 
    CHECK(hipFree(atom->g_NL))
    CHECK(hipFree(atom->g_x))
    CHECK(hipFree(atom->g_y))
    CHECK(hipFree(atom->g_z))
    CHECK(hipFree(atom->g_fx))
    CHECK(hipFree(atom->g_fy))
    CHECK(hipFree(atom->g_fz))
    CHECK(hipFree(atom->g_pe))
    CHECK(hipFree(atom->g_box))
}

