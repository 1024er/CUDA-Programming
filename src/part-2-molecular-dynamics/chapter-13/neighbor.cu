#include "hip/hip_runtime.h"
#include "neighbor.h"
#include "mic.h"
#include <stdio.h>
#include <stdlib.h>

static void __global__ gpu_find_neighbor
(
    int N, int *g_NN, int *g_NL, double *g_box, 
    double *g_x, double *g_y, double *g_z, double cutoff2
)
{
    int n1 = blockIdx.x * blockDim.x + threadIdx.x;
    if (n1 < N)
    {
        int count = 0;
        double x1 = g_x[n1];
        double y1 = g_y[n1];
        double z1 = g_z[n1];
        for (int n2 = 0; n2 < N; n2++)
        {
            double x12 = g_x[n2] - x1;
            double y12 = g_y[n2] - y1;
            double z12 = g_z[n2] - z1;
            apply_mic(g_box, &x12, &y12, &z12);
            double d12_square = x12*x12 + y12*y12 + z12*z12;
            if ((n2 != n1) && (d12_square < cutoff2))
            {
                g_NL[count++ * N + n1] = n2;
            }
        }
        g_NN[n1] = count;
    }
}

void find_neighbor(int N, int MN, Atom *atom)
{
    double cutoff = 10.0;
    double cutoff2 = cutoff * cutoff;

    int block_size = 128;
    int grid_size = (N - 1) / block_size + 1;
    gpu_find_neighbor<<<grid_size, block_size>>>
    (
        N, atom->g_NN, atom->g_NL, atom->g_box,
        atom->g_x, atom->g_y, atom->g_z, cutoff2
    );
}

